
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_kernel() {
    // calculate global thread identifier, note blockIdx.x = 0 here
    const int thid = blockDim.x * blockIdx.x + threadIdx.x;

    // print a greeting message
    printf("Hello from thread %d\n", thid);
}


int main(int argc, char* argv[]) {

    // set the ID of the CUDA device
    hipSetDevice(0);

    // invoke kernel using 4 threads executed in 1 thread block
    hello_kernel<<<1, 4>>>();

    // synchronize the GPU preventing premature termination
    hipDeviceSynchronize();
    
}
