
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>       /* time manipulation */
#include <iomanip>      /* float number precission */

const int LENGTH{1 << 30};
const int MAX_VAL{1 << 5};

/**
 * @description: add up vector a and vector b with CUDA
 * @param {float*} a: matrix A 
 * @param {float*} b: matrix B
 * @param {float*} result: the result of vector addition
 * @param {int} length: the length of the vector
 */
__global__ void add_vectors(float* a, float* b, float* result, int length) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tid < length ) {
        result[tid] = a[tid] + b[tid];
    }

}

/**
 * @description: add up vector a and vector b with CUDA
 * @param {float*} a: matrix A 
 * @param {float*} b: matrix B
 * @param {float*} result: the result of vector addition
 * @param {float*} result_from_cuda: the result of vector addition
 * @param {int} length: the length of the vector
 */
void initialize(float* a, float* b, float* result, float* result_from_cuda, int length) {
    srand(1);
    for ( int i = 0; i < length; i++ ) {
        a[i] = rand() % MAX_VAL / 100.0;
        b[i] = rand() % MAX_VAL / 100.0;
        result[i] = 0;
        result_from_cuda[i] = 0;
    }
}

/**
 * @description: sequential version of adding up vector a and vector b
 * @param {float*} a: matrix A 
 * @param {float*} b: matrix B
 * @param {float*} result: the result of vector addition
 * @param {int} length: the length of the vector
 */
void sequential_add_vectors(float* a, float* b, float* result, int length) {
    for ( int i = 0; i < length; i++ ) {
        result[i] = a[i] + b[i];
    }
}


bool validate_result(float* a, float* b, int length) {
    
    std::cout << "Validating result ... " << std::endl;

    for ( int i = 0; i < length; i++ ) {
        if ( a[i] != b[i] ) {
            std::cout << "false" << std::endl;
            return false;
        }
    }
    
    std::cout << "true" << std::endl;
    return true;
}


int main() {

    // create local data objects
    float* a = new float[LENGTH];
    float* b = new float[LENGTH];
    float* result = new float[LENGTH];
    float* result_from_cuda = new float[LENGTH];

    initialize(a, b, result, result_from_cuda, LENGTH);

    // time manipulation
    std::chrono::steady_clock::time_point start_time;
    std::chrono::steady_clock::time_point end_time;
    std::chrono::steady_clock::time_point preprocess_end_time;
    std::chrono::steady_clock::time_point cleanup_end_time;
    std::chrono::duration<double> duration;
    std::chrono::duration<double> preprocess_time;
    std::chrono::duration<double> cleanup_time;

    double t1;
    double tp;
    double t_preprocess;
    double t_cleanup;

    // sequential version
    std::cout << "============================================" << std::endl;
    std::cout << "Running sequential version: " << std::endl;
    start_time = std::chrono::steady_clock::now();
    sequential_add_vectors(a, b, result, LENGTH);
    end_time = std::chrono::steady_clock::now();
    duration = end_time - start_time;
    t1 = duration.count();

    std::cout << std::setprecision(2) << "it takes " << t1 << " seconds." << std::fixed << std::scientific << std::endl;
    std::cout << "============================================" << std::endl;

    // cuda version
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (LENGTH + threadsPerBlock - 1) / threadsPerBlock;
    float* cuda_a;
    float* cuda_b;
    float* cuda_result;

    std::cout << "Running cuda version: " << std::endl;
    start_time = std::chrono::steady_clock::now();

    // set the ID of the CUDA device
    hipSetDevice(0);

    // Allocate vectors in device memory
    hipMalloc(&cuda_a, sizeof(float) * LENGTH);
    hipMalloc(&cuda_b, sizeof(float) * LENGTH);
    hipMalloc(&cuda_result, sizeof(float) * LENGTH);

    // copy vectors from host memory to device memory
    hipMemcpy(cuda_a, a, sizeof(float) * LENGTH, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof(float) * LENGTH, hipMemcpyHostToDevice);
    hipMemcpy(cuda_result, result_from_cuda, sizeof(float) * LENGTH, hipMemcpyHostToDevice);
    preprocess_end_time = std::chrono::steady_clock::now();
    
    // invoke kernel
    add_vectors<<<blocksPerGrid, threadsPerBlock>>>(cuda_a, cuda_b, cuda_result, LENGTH);
    // copy result from device memory to host memeoy
    hipMemcpy(result_from_cuda, cuda_result, sizeof(float) * LENGTH, hipMemcpyDeviceToHost);

    end_time = std::chrono::steady_clock::now();
    duration = end_time - preprocess_end_time;

    // Free device memory
    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_result);
    cleanup_end_time = std::chrono::steady_clock::now();

    // synchronize the GPU preventing premature termination
    hipDeviceSynchronize();

    tp = duration.count();
    preprocess_time = preprocess_end_time - start_time;
    t_preprocess = preprocess_time.count();
    cleanup_time = cleanup_end_time - end_time;
    t_cleanup = cleanup_time.count();

    std::cout << "it takes " << std::setprecision(2)  << tp << " seconds." << std::scientific << std::endl;
    std::cout << "preprocess time: " << std::setprecision(2)  << t_preprocess  << std::scientific << " seconds." << std::endl;
    std::cout << "cleanup_time: " << std::setprecision(2)  << t_cleanup << std::scientific << " seconds." << std::endl;
    std::cout << "total: " << std::setprecision(2)  << tp + t_preprocess + t_cleanup << std::scientific << " seconds." << std::endl;
    std::cout << "============================================" << std::endl;

    // validating result
    validate_result(result, result_from_cuda, LENGTH);

    // cleanup host memory
    delete[] a;
    delete[] b;
    delete[] result;
    delete[] result_from_cuda;

    std::cout << "============================================" << std::endl;

    // calculate speed up
    double speedup = t1 / tp;
    std::cout << "speedup: " << speedup << std::scientific << std::endl;

    // efficiency
    double efficiency = speedup / blocksPerGrid / threadsPerBlock;
    std::cout << "efficiency: " << efficiency << std::scientific << std::endl;

    std::cout << "============================================" << std::endl;
    std::cout << "if we take the preprocess time and the cleanup time as a cost of cuda computation" << std::endl;
    speedup = t1 / (tp + t_preprocess + t_cleanup);
    std::cout << "speedup: " << speedup << std::scientific << std::endl;
    efficiency = speedup / blocksPerGrid / threadsPerBlock;
    std::cout << "efficiency: " << efficiency << std::scientific << std::endl;
    std::cout << "============================================" << std::endl;

    return 0;
}

