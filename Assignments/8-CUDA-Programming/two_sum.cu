
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>       /* time manipulation */

const int LENGTH{1 << 30};
const int MAX_VAL{1 << 4};

/**
 * @description: kernel definition
 * @param {float*} x: vector x
 * @param {float*} y: vector y
 * @param {float*} z: vector z, the addition of x and y
 * @param {int} length: the length of vectors
 */
__global__ void add_vectors(float* x, float* y, float* z, int length) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tid < length ) {
        z[tid] = x[tid] + y[tid];
    }

}

/**
 * @description: initialize vectors
 * @param {float*} x: vector x
 * @param {float*} y: vector y
 * @param {float*} z: vector z
 * @param {float*} z_from_cuda: vector z_from_cuda
 * @param {int} length: the length of vectors
 */
void initialize(float* x, float* y, float* z, float* z_from_cuda, int length) {
    srand(1);
    for ( int i = 0; i < length; i++ ) {
        x[i] = rand() % MAX_VAL;
        y[i] = rand() % MAX_VAL;
        z[i] = 0;
        z_from_cuda[i] = 0;
    }
}

/**
 * @description: sequential version of vector addition
 * @param {float*} x: vector x
 * @param {float*} y: vector y
 * @param {float*} z: vector z
 * @param {int} length: the length of vectors
 */
void sequential_add_vectors(float* x, float* y, float* z, int length) {
    for ( int i = 0; i < length; i++ ) {
        z[i] = x[i] + y[i];
    }
}

/**
 * @description: validate results from sequential version and parallel version
 * @param {float*} h_z: vector h_z
 * @param {float*} h_z_from_cuda: vector h_z_from_cuda
 * @param {int} length: the length of vectors
 */
void validate_result(float* h_z, float* h_z_from_cuda, int length){
    for ( int i = 0; i < length; i++ ) {
        if ( h_z_from_cuda[i] != h_z[i] ){
            std::cout << "False" << std::endl;
            return;
        }
    }
    
    std::cout << "True" << std::endl;
}


int main() {
    
    std::chrono::steady_clock::time_point start_time_cuda;
    std::chrono::steady_clock::time_point end_time_cuda;
    std::chrono::duration<double> duration_cuda;
    std::chrono::steady_clock::time_point start_time;
    std::chrono::steady_clock::time_point end_time;
    std::chrono::duration<double> duration;
    double t1;
    double tp;

     // set the ID of the CUDA device
    hipSetDevice(0);

    // create local data objects
    float* h_x = new float[LENGTH];
    float* h_y = new float[LENGTH];
    float* h_z = new float[LENGTH];
    float* h_z_from_cuda = new float[LENGTH];

    initialize(h_x, h_y, h_z, h_z_from_cuda, LENGTH);

    std::cout << "Running sequential version: " << std::endl;
    start_time = std::chrono::steady_clock::now();

    sequential_add_vectors(h_x, h_y, h_z, LENGTH);

    end_time = std::chrono::steady_clock::now();
    duration = end_time - start_time; 
    t1 = duration.count();
    std::cout << "Time: " << t1 << std::endl;
   
    // cuda version
    const int threadsPerBlock = 1024;
    const int numOfBlocks = (LENGTH + threadsPerBlock - 1) / threadsPerBlock;

    size_t size = LENGTH * sizeof(float);

    float* d_x;
    float* d_y;
    float* d_z;

    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_z, size);

    
    std::cout << "Running cuda version: " << std::endl;
    start_time_cuda = std::chrono::steady_clock::now();

    // Copy vectors from host memory to device memory
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

    // cuda operation
    add_vectors<<<numOfBlocks, threadsPerBlock>>>(d_x, d_y, d_z, LENGTH);

    // Copy result from device memory to host memory
    hipMemcpy(h_z_from_cuda, d_z, size, hipMemcpyDeviceToHost);

    end_time_cuda = std::chrono::steady_clock::now();
    duration_cuda = end_time_cuda - start_time_cuda;
    tp = duration_cuda.count();
    std::cout << "Time: " << tp << std::endl;

    // free device memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    
    // synchronize the GPU preventing premature termination
    hipDeviceSynchronize();

    validate_result(h_z, h_z_from_cuda, LENGTH);

    // free host memory
    delete[] h_z_from_cuda;
    delete[] h_z;
    delete[] h_x;
    delete[] h_y;

    double speedup = t1 / tp;
    double efficiency = speedup / (numOfBlocks * threadsPerBlock);
    std::cout << "speedup: " << speedup << std::endl;
    std::cout << "efficiency: " << efficiency << std::endl;

    return 0;
}

